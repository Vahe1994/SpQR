#include "hip/hip_runtime.h"
/*
 * Copyright (C) SPQR Kernel.2024 Elvir Crncevic (elvircrn@gmail.com)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common.cuh"
#include <ATen/cuda/Exceptions.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cuda_pipeline.h>

#define DEVICE_INLINE __forceinline__ __device__


extern "C" __device__ uint32_t __nvvm_get_smem_pointer(void *);


template<class Acc_t> constexpr __device__ __host__ bool is_fp32() {
  if constexpr (std::is_same_v<Acc_t, float> || std::is_same_v<Acc_t, float2>) {
    return true;
  }
  return false;
}

DEVICE_INLINE uint64_t recover_second_order(uint64_t val) {
  constexpr unsigned int FULL_MASK = 0xffffffffu;
  val |= __shfl_xor_sync(FULL_MASK, val, 2);
  val |= __shfl_xor_sync(FULL_MASK, val, 4);
  val |= __shfl_xor_sync(FULL_MASK, val, 8);
  return val;
}

using u64 = unsigned long long;
using s32 = int;
using u32 = unsigned int;
using u16 = unsigned short;

union RowBits {
  uint64_t mask;

  struct {
    uint64_t s: 3;
    uint64_t z: 3;
    uint64_t w: 48;
  };

  __device__ __forceinline__ u16 get_w(int i) const {
    return (w >> (i * 3u)) & ((1u << 3u) - 1u);
  }

  __device__ __forceinline__ u32 get_w2(int i) const {
    return (mask >> (i * 6u)) & ((1u << 6u) - 1u);
  }
};

half2 DEVICE_INLINE dequantize2(const half2 &q,
                                const half2 &s,
                                const half2 &z) {
  const half2 &res = __hmul2(s, __hsub2(q, z));
  return res;
}

template<class Bit_t, class Scalar_t> DEVICE_INLINE Scalar_t dequantize(Bit_t q,
                                                                        Scalar_t s,
                                                                        Scalar_t z) {
  if constexpr (std::is_same<Bit_t, half>::value) {
    return __hmul(s, __hsub(q, z));
  } else {
    return __hmul(s, __hsub(__uint2half_rd(q, z)));
  }
}

#define CUINLINE __forceinline__

#define UPDIV(X, Y) (((X) + (Y)-1) / (Y))

[[nodiscard]] __device__ __host__ CUINLINE int updiv(int x, int y) {
  return (x + y - 1) / y;
}

struct Timer {
  hipEvent_t ce_start{}, ce_stop{};
  hipStream_t stream;

  void start() { AT_CUDA_CHECK(hipEventRecord(ce_start, stream)); }

  float end() {
    float time;
    AT_CUDA_CHECK(hipEventRecord(ce_stop, 0));
    AT_CUDA_CHECK(hipEventSynchronize(ce_stop));
    AT_CUDA_CHECK(hipEventElapsedTime(&time, ce_start, ce_stop));
    // Returns ms
    return time;
  }

  Timer(hipStream_t stream) : stream(stream) {
    AT_CUDA_CHECK(hipEventCreate(&ce_start));
    AT_CUDA_CHECK(hipEventCreate(&ce_stop));
  }

  Timer(Timer &&timer) = delete;

  Timer(const Timer &timer) = delete;

  ~Timer() {
    AT_CUDA_CHECK(hipEventDestroy(ce_start));
    AT_CUDA_CHECK(hipEventDestroy(ce_stop));
  }
};


template<typename T> __device__ T _debug_halfs(T v) {
  if constexpr (std::is_same<T, half>::value) {
    printf(" %f\n", __half2float(v));
  } else if constexpr (std::is_same<T, half2>::value) {
    printf(" %f %f\n", __half2float(v.x), __half2float(v.y));
  }
  return v;
}

template<typename T, typename... Arguments> __device__ void _debug_halfs(T v, Arguments... vals) {
  if constexpr (std::is_same<T, half>::value) {
    printf(" %f", __half2float(v));
  } else if constexpr (std::is_same<T, half2>::value) {
    printf(" %f %f", __half2float(v.x), __half2float(v.y));
  }
  _debug_halfs(vals...);
}

template<class Scalar_t> __host__ __device__ auto vectorize(Scalar_t *ptr) {
  if constexpr (std::is_same<Scalar_t, float>::value) {
    return reinterpret_cast<float2 *>(ptr);
  } else if constexpr (std::is_same<Scalar_t, half>::value) {
    return reinterpret_cast<half2 *>(ptr);
  } else {
    return ptr;
  }
}

template<class Vec_t> __host__ __device__ auto scalarize(void *ptr) {
  if constexpr (std::is_same<Vec_t, float>::value ||
                std::is_same<Vec_t, float2>::value) {
    return reinterpret_cast<float *>(ptr);
  } else if constexpr (std::is_same<Vec_t, half2>::value) {
    return reinterpret_cast<half *>(ptr);
  } else {
    return ptr;
  }
}

DEVICE_INLINE float add_and_accum(float a, float b) {
  return a + b;
}

DEVICE_INLINE half add_and_accum(const half2 &a, const half2 &b) {
  half2 r = __hadd2(a, b);
  return __hadd(r.x, r.y);
}

template<class T> DEVICE_INLINE u16 get_col(T m) {
  return static_cast<u16>(m & T((1u << 16u) - 1u));
}

DEVICE_INLINE half get_val(u32 m) {
  u16 _v = m >> 16u;
  half v = *reinterpret_cast<half *>(&_v);
  return v;
}


#define CALL_DENSE(F, _BLOCK_HEIGHT, _BLOCK_WIDTH, PIPELINE_DEPTH) \
    constexpr int BLOCK_HEIGHT = _BLOCK_HEIGHT; \
    constexpr int BLOCK_WIDTH = _BLOCK_WIDTH; \
    size_t smem_size = sizeof(half2) * (BLOCK_WIDTH * SHARED_OFFSET);                   \
    F<3, 16, 16, BLOCK_HEIGHT, BLOCK_WIDTH, float, uint64_t, PIPELINE_DEPTH> \
            <<<dim3(updiv(prob_m, 16 * BLOCK_HEIGHT), 1, 1), \
            dim3(__min(updiv(prob_n, 16), BLOCK_WIDTH) * 16, 1, 1), smem_size, \
            stream>>>(prob_m, \
            prob_n, \
            raw_data,                               \
            X_ptr, \
            order_ptr, \
            y_ptr);


#define CALL_FUSED(F, _BLOCK_HEIGHT, _BLOCK_WIDTH, PIPELINE_DEPTH) \
    constexpr int BLOCK_HEIGHT = _BLOCK_HEIGHT; \
    constexpr int BLOCK_WIDTH = _BLOCK_WIDTH; \
    size_t smem_size = sizeof(half2) * prob_n / 2;                   \
    F<3, 16, 16, BLOCK_HEIGHT, BLOCK_WIDTH, float, uint64_t, PIPELINE_DEPTH> \
            <<<dim3(updiv(prob_m, 16 * BLOCK_HEIGHT), 1, 1), \
            dim3(__min(updiv(prob_n, 16), BLOCK_WIDTH) * 16, 1, 1), smem_size, \
            stream>>>(prob_m, \
            prob_n, \
            raw_data,                               \
            X_ptr, \
            row_offsets_ptr, \
            col_vals_ptr, \
            order_ptr, \
            y_ptr);


static constexpr u32 SHARED_OFFSET = 32;

// Wait until at most `n` async copy stages are still pending.
template<int n> DEVICE_INLINE void cp_async_wait() {
  asm volatile("cp.async.wait_group %0;\n"::"n"(n));
}

template<int BITS, int BETA1, int BETA2, int BLOCK_HEIGHT, int BLOCK_WIDTH, class Acc_t, class W_t /* = uint64_t */, int
  PIPELINE_DEPTH> __global__ void spqr_quantized_matvec_dense(
  // W and meta
  unsigned int prob_m,
  unsigned int prob_n,
  // W 1st order stats
  const W_t *__restrict__ raw_data,
  const half *__restrict__ x,
  // Outliers
  const short *__restrict__ order,
  // Output
  half *__restrict__ y_fp16) {
  /*
           ┌─────────────┐ ┌─┐   ┌─┐
   beta1   │   block 0   │ │ │   │ │
           ├─────────────┤ │ │   │ │
   beta1   │   block 1   │ │ │   │ │
           └─────────────┘ │x│ = │Y│
           │    ...      │ │ │   │ │
           ┌─────────────┐ │ │   │ │
   beta1   │  block m-1  │ │ │   │ │
           └─────────────┘ └─┘   └─┘
  */
  static constexpr int WARP_SIZE = 32;

  extern __shared__ half2 s_x2[];
  __shared__ half2 s_half2_lut_global[64 * BLOCK_WIDTH];
  __shared__ Acc_t s_y[BETA1];

  static constexpr int HALF_WARP_SIZE = 16;
  auto s_half2_lut = s_half2_lut_global + ((threadIdx.x / HALF_WARP_SIZE) << 6);

#pragma loop unroll
  for (int i = threadIdx.x % HALF_WARP_SIZE; i < 64; i += HALF_WARP_SIZE) {
    s_half2_lut[i] = make_half2(
      __int2half_rd(i & 0b111),
      __int2half_rd(i >> 3));
  }

  const half2 *x2 = reinterpret_cast<const half2 *>(x);


  if constexpr (std::is_same<Acc_t, float>::value) {
    if (threadIdx.x < BETA1) {
      // TOD: Check if this really sets s_y to zero.
      asm volatile ("cp.async.ca.shared.global [%0], [%0], 4, 0 ;\n" :
        : "r"(__nvvm_get_smem_pointer(s_y + threadIdx.x))
      );
    }
  } else {
    if (threadIdx.x < BETA1 / 2) {
      asm volatile ("cp.async.ca.shared.global [%0], [%0], 4, 0 ;\n" :
        : "r"(__nvvm_get_smem_pointer(s_y + threadIdx.x))
      );
    }
  }

  asm volatile ("cp.async.commit_group;");
  constexpr u32 THREAD_COUNT = BLOCK_WIDTH * BETA1; // = 128 (example)

  // Number of SPQR tiles that this CUDA block will process.
  u32 num_spqr_tiles_per_cuda_block = UPDIV(prob_n, BETA2);

  // Here is how we organize things here. We have THREAD_COUNT threads in a
  // block in x-dimension. We distribute 1 thread per tile row. Therefore, we
  // have BETA1 threads per tile. For now, a block only spans across 1 dimension
  // of SPQR tiles.
  constexpr u32 NUM_SPQR_TILES_PER_ITERATION = THREAD_COUNT / BETA1;

  u32 row_pos = threadIdx.x & 0xF; // threadIdx.x % BETA1;
  const u32 subtile_id = threadIdx.x / BETA1;

  const W_t *local_raw_data =
      raw_data + blockIdx.x * num_spqr_tiles_per_cuda_block * BETA1 + subtile_id * BETA1 + row_pos;

  constexpr u32 FULL_MASK = 0xffffffff;
  constexpr u32 HALF_MASK = FULL_MASK >> 16u;

  constexpr static unsigned long long int NUM_USEFUL_BITS = 18ull * static_cast<u64>(BITS);
  constexpr static int OFFSET = BETA1 / SECOND_ORDER_FRAGMENT_SIZE_BITS;

  const auto s_x2_ = s_x2 + subtile_id * SHARED_OFFSET;


  cp_async_wait<0>();
  Acc_t acc{};
  __syncthreads();
  for (u32 i = subtile_id; i < num_spqr_tiles_per_cuda_block; i += NUM_SPQR_TILES_PER_ITERATION, local_raw_data +=
                                                              NUM_SPQR_TILES_PER_ITERATION *
                                                              BETA1) {
#if 0
    asm volatile ("cp.async.ca.shared.global [%0], [%1], 4 ;\n"::"r"(__nvvm_get_smem_pointer(s_x2 + subtile_id * SHARED_OFFSET + (threadIdx.x & 0xF) / 2)), "l"(x2 + i * BETA2 / 2 + (threadIdx.x & 0xF) / 2));
    asm volatile ("cp.async.commit_group;");
#else
    s_x2[subtile_id * SHARED_OFFSET + (threadIdx.x & 0xF) / 2] = x2[i * BETA2 / 2 + (threadIdx.x & 0xF) / 2];
#endif

    auto v = __ldg(local_raw_data);
    RowBits row_bits{
      .mask = v
    };
    uint64_t s_order_partial =
        (row_bits.mask >> NUM_USEFUL_BITS) << (SECOND_ORDER_FRAGMENT_SIZE_BITS * (row_pos / OFFSET));
    SecondOrder _s{.v = recover_second_order(s_order_partial)};


    half2 first_order_quantized = s_half2_lut[row_bits.get_w2(0)];
    half2 first_order_dequantized = dequantize2(first_order_quantized,
                                                _s.get_sws2(),
                                                _s.get_swz2());

    half2 ws2 = __half2half2(first_order_dequantized.x);
    half2 wz2 = __half2half2(first_order_dequantized.y);

#if 0
    cp_async_wait<0>();
#else
    __threadfence_block();
#endif

#pragma unroll
    for (u32 j = 0; j < BETA2 / 2; j++) {
      if constexpr (std::is_same<Acc_t, float>::value) {
        half2 q = s_half2_lut[row_bits.get_w2(j + 1)];
        half2 w = dequantize2(q, ws2, wz2);
        float2 x_fp32 = __half22float2(s_x2_[j]);
        float2 w_fp32 = __half22float2(w);
        acc = fmaf(x_fp32.x, w_fp32.x, acc);
        acc = fmaf(x_fp32.y, w_fp32.y, acc);
      } else {
        int q_x = row_bits.get_w(2 * j);
        int q_y = row_bits.get_w(2 * j + 1);
        half2 q = make_half2(__int2half_rd(q_x), __int2half_rd(q_y));
        half2 w = dequantize2(q, ws2, wz2);
        acc = __hfma2(s_x2[i * BETA2 / 2 + j], w, acc);
      }
    }
  }

  auto s_y_scalar = scalarize<Acc_t>(s_y);
  auto s_y_vectorized = vectorize(s_y_scalar);

  auto other = __shfl_down_sync(HALF_MASK, acc, BETA1);
  auto result = add_and_accum(other, acc);
  const unsigned int lane_id = threadIdx.x & 0x1F;
  if constexpr (std::is_same_v<Acc_t, float>) {
    if (lane_id < BETA1) {
      atomicAdd(s_y_scalar + lane_id, result);
    }
  } else {
    auto result0 = __shfl_down_sync(0, result, threadIdx.x);
    auto result1 = __shfl_down_sync(0, result, threadIdx.x + 1);
    if (lane_id < BETA1 / 2) {
      atomicAdd(s_y_vectorized + lane_id, make_half2(result0, result1));
    }
  }

  __syncthreads();

  if (order == nullptr) {
    if (threadIdx.x < BETA1 / 2) {
      reinterpret_cast<half2 *>(y_fp16)[blockIdx.x * (BETA1 / 2) +
                                        threadIdx.x] = __float22half2_rn(s_y_vectorized[threadIdx.x]);
    }
  } else {
    if (threadIdx.x < BETA1) {
      short row = order[blockIdx.x * BETA1 + threadIdx.x];
      y_fp16[row] = __float2half(s_y_scalar[threadIdx.x]);
    }
  }
}


template<int BITS, int BETA1, int BETA2, int BLOCK_HEIGHT, int BLOCK_WIDTH, class Acc_t, class W_t /* = uint64_t */, int
  PIPELINE_DEPTH> __global__ void spqr_quantized_matvec_fused(
  // W and meta
  unsigned int prob_m,
  unsigned int prob_n,
  // W 1st order stats
  const W_t *__restrict__ raw_data,
  const half *__restrict__ x,
  // Outliers
  const int *__restrict__ row_offsets,
  const u32 *__restrict__ col_vals,
  const short *__restrict__ order,
  // Output
  half *__restrict__ y_fp16) {
  /*
           ┌─────────────┐ ┌─┐   ┌─┐
   beta1   │   block 0   │ │ │   │ │
           ├─────────────┤ │ │   │ │
   beta1   │   block 1   │ │ │   │ │
           └─────────────┘ │x│ = │Y│
           │    ...      │ │ │   │ │
           ┌─────────────┐ │ │   │ │
   beta1   │  block m-1  │ │ │   │ │
           └─────────────┘ └─┘   └─┘
  */
  static constexpr u32 WARP_SIZE = 32;
  static constexpr u32 NUM_HALF_WARPS = BLOCK_HEIGHT * BLOCK_WIDTH;
  static constexpr u32 THREAD_COUNT = BLOCK_HEIGHT * BLOCK_WIDTH * WARP_SIZE / 2;
  static constexpr u32 OUTPUT_SIZE = BETA1 * BLOCK_HEIGHT;
#if 0
  extern __shared__ half2 s_x2[];
  __shared__ half2 s_half2_lut_global[64 * BLOCK_WIDTH];
  __shared__ Acc_t s_y[BETA1];
  __shared__ u32 s_row_offsets[BETA1 + 1];

  static constexpr int HALF_WARP_SIZE = 16;
  auto s_half2_lut = s_half2_lut_global + ((threadIdx.x / HALF_WARP_SIZE) << 6);

#pragma loop unroll
  for (int i = threadIdx.x % HALF_WARP_SIZE; i < 64; i += HALF_WARP_SIZE) {
    s_half2_lut[i] = make_half2(
        __int2half_rd(i & 0b111),
        __int2half_rd(i >> 3)
    );
  }

  const half2 *x2 = reinterpret_cast<const half2 *>(x);

  u32 pipeline_depth{};

  const auto total_threads = blockDim.x;
  const auto x2_count = prob_n / 2;
  const auto tid = threadIdx.x;
  u32 pipeline_id{};
#else
  extern __shared__ half2 s_x2[];
  __shared__ half2 s_half2_lut_global[64 * NUM_HALF_WARPS];
  __shared__ Acc_t s_y[OUTPUT_SIZE];
  __shared__ u32 s_row_offsets[OUTPUT_SIZE + 1];

  const u32 thread_xy = threadIdx.x + (threadIdx.y * blockDim.x);

  static constexpr u32 HALF_WARP_SIZE = 16;

  for (u32 i = thread_xy; i < 64 * NUM_HALF_WARPS; i += THREAD_COUNT) {
    s_half2_lut_global[i] = make_half2(
      __int2half_rd(i & 0b111),
      __int2half_rd((i >> 3u) & 0b111)
    );
  }

  auto s_half2_lut = s_half2_lut_global + ((thread_xy / HALF_WARP_SIZE) << 6);
  const half2 *x2 = reinterpret_cast<const half2 *>(x);

  const auto total_threads = blockDim.x;
  const auto x2_count = prob_n / 2;
  const auto tid = threadIdx.x;
#endif


  if constexpr (std::is_same<Acc_t, float>::value) {
    if (threadIdx.x < BETA1) {
      // TOD: Check if this really sets s_y to zero.
      asm volatile ("cp.async.ca.shared.global [%0], [%0], 4, 0 ;\n" :
        : "r"(__nvvm_get_smem_pointer(s_y + threadIdx.x))
      );
    }
  } else {
    if (threadIdx.x < BETA1 / 2) {
      asm volatile ("cp.async.ca.shared.global [%0], [%0], 4, 0 ;\n" :
        : "r"(__nvvm_get_smem_pointer(s_y + threadIdx.x))
      );
    }
  }

  // Here we load the row offsets into smem.
  for (int i = threadIdx.x; i <= BETA1; i += blockDim.x) {
    __pipeline_memcpy_async(s_row_offsets + i, row_offsets + blockIdx.x * BETA1 + i, sizeof(u32));
  }

  u32 idx = tid;
  u32 pipeline_id{};
  u32 pipeline_stack_ptr{};
  for (; pipeline_id < PIPELINE_DEPTH && idx < x2_count; pipeline_id++, idx += THREAD_COUNT) {
    __pipeline_memcpy_async(s_x2 + idx, x2 + idx, sizeof(half2));
    pipeline_stack_ptr++;
    __pipeline_commit();
  }


  const u32 blockId = blockIdx.x;

  // Number of SPQR tiles that this CUDA block will process.
  u32 num_spqr_tiles_per_cuda_block = UPDIV(prob_n, BETA2);

  // Here is how we organize things here. We have THREAD_COUNT threads in a
  // block in x-dimension. We distribute 1 thread per tile row. Therefore, we
  // have BETA1 threads per tile. For now, a block only spans across 1 dimension
  // of SPQR tiles.
  constexpr u32 NUM_SPQR_TILES_PER_ITEARTION = THREAD_COUNT / BETA1;

  u32 row_pos = threadIdx.x & 0xF; // threadIdx.x % BETA1;
  const u32 subtile_id = threadIdx.x / BETA1;

  if (subtile_id >= UPDIV(prob_n, BETA2)) {
    return;
  }

  const W_t *local_raw_data =
      raw_data + blockIdx.x * num_spqr_tiles_per_cuda_block * BETA1 + subtile_id * BETA1 + row_pos;

  constexpr u32 FULL_MASK = 0xffffffff;
  constexpr u32 HALF_MASK = FULL_MASK >> 16u;

  if ((row_pos + blockId * BETA1) >= prob_m) {
    // TODO: Maybe don't do this, since we need these threads to load x
    // together? [1]
    return;
  } // || (threadIdx.x % BETA1)


  constexpr static unsigned long long int NUM_USEFUL_BITS = 18ull * static_cast<u64>(BITS);
  constexpr static int OFFSET = BETA1 / SECOND_ORDER_FRAGMENT_SIZE_BITS;

  Acc_t acc{};
  for (u32 i = subtile_id; i < num_spqr_tiles_per_cuda_block;
       i += NUM_SPQR_TILES_PER_ITEARTION, local_raw_data += NUM_SPQR_TILES_PER_ITEARTION * BETA1) {
    auto v = __ldg(local_raw_data);
    RowBits row_bits{
      .mask = v
    };
    uint64_t s_order_partial =
        (row_bits.mask >> NUM_USEFUL_BITS) << (SECOND_ORDER_FRAGMENT_SIZE_BITS * (row_pos / OFFSET));
    SecondOrder _s{.v = recover_second_order(s_order_partial)};
    half2 first_order_quantized = s_half2_lut[row_bits.get_w2(0)];
    half2 first_order_dequantized = dequantize2(first_order_quantized,
                                                _s.get_sws2(),
                                                _s.get_swz2());

    half2 ws2 = __half2half2(first_order_dequantized.x);
    half2 wz2 = __half2half2(first_order_dequantized.y);

    const auto s_x2_ = s_x2 + i * (BETA2 >> 1);

    if (pipeline_stack_ptr > 0) {
      __pipeline_wait_prior(pipeline_stack_ptr - 1);
      pipeline_stack_ptr--;
    }

    __syncthreads();
#pragma unroll
    for (u32 j = 0; j < BETA2 / 2; j++) {
      if constexpr (std::is_same<Acc_t, float>::value) {
        half2 q = s_half2_lut[row_bits.get_w2(j + 1)];
        half2 w = dequantize2(q, ws2, wz2);
        float2 x_fp32 = __half22float2(s_x2_[j]);
        float2 w_fp32 = __half22float2(w);
        acc = fmaf(x_fp32.x, w_fp32.x, acc);
        acc = fmaf(x_fp32.y, w_fp32.y, acc);
      } else {
        int q_x = row_bits.get_w(2 * j);
        int q_y = row_bits.get_w(2 * j + 1);
        half2 q = make_half2(__int2half_rd(q_x), __int2half_rd(q_y));
        half2 w = dequantize2(q, ws2, wz2);
        acc = __hfma2(s_x2[i * BETA2 / 2 + j], w, acc);
      }
    }

    unsigned idx = pipeline_id * total_threads + tid;
    if (idx < prob_n / 2) {
      __pipeline_memcpy_async(s_x2 + idx, x2 + idx, sizeof(half2));
      pipeline_id++;
      pipeline_stack_ptr++;
      __pipeline_commit();
    }
  }

  auto s_y_scalar = scalarize<Acc_t>(s_y);
  auto s_y_vectorized = vectorize(s_y_scalar);

  u32 t = row_pos;
  u32 s = s_row_offsets[t];
  u32 e = s_row_offsets[t + 1];
  u32 wid = subtile_id;

  half *s_x = reinterpret_cast<half *>(s_x2);

#if 1
  for (u32 i = s + wid; i < e; i += BLOCK_WIDTH) {
    ColVal colval{
      ._ = __ldg(col_vals + i)
    };
    auto c = colval.members.c;
    auto v = colval.members.v;
    acc += __half2float(v) * __half2float(s_x[c]);
  }
#endif

  auto other = __shfl_down_sync(HALF_MASK, acc, BETA1);
  auto result = add_and_accum(other, acc);
  const unsigned int lane_id = threadIdx.x & 0x1F;
  if constexpr (std::is_same_v<Acc_t, float>) {
    if (lane_id < BETA1) {
      atomicAdd(s_y_scalar + lane_id, result);
    }
  } else {
    auto result0 = __shfl_down_sync(0, result, threadIdx.x);
    auto result1 = __shfl_down_sync(0, result, threadIdx.x + 1);
    if (lane_id < BETA1 / 2) {
      atomicAdd(s_y_vectorized + lane_id, make_half2(result0, result1));
    }
  }

  __syncthreads();

  if (order == nullptr) {
    if (threadIdx.x < BETA1 / 2) {
      reinterpret_cast<half2 *>(y_fp16)[blockIdx.x * (BETA1 / 2) +
                                        threadIdx.x] = __float22half2_rn(s_y_vectorized[threadIdx.x]);
    }
  } else {
    if (threadIdx.x < BETA1) {
      short row = order[blockIdx.x * BETA1 + threadIdx.x];
      y_fp16[row] = __float2half(s_y_scalar[threadIdx.x]);
    }
  }
}

template<class T> const T &__min(const T &a, const T &b) {
  return (b < a) ? b : a;
}

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,     \
             hipGetErrorString(status), status);                              \
      return EXIT_FAILURE;                                                     \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, \
             hipsparseGetErrorString(status), status);                          \
      return EXIT_FAILURE;                                                     \
    }                                                                          \
  }

union Features {
  uint32_t _;

  struct {
    uint32_t is_fp32: 1;
    uint32_t dense_only: 1;
    uint32_t naive_sparse: 1;
    uint32_t torch: 1;
    uint32_t is_async: 1;
    uint32_t shared_sparse: 1;
    uint32_t single_sparse: 1;
    uint32_t cusparse: 1;
    uint32_t fused_sparse: 1;
    uint32_t shared_sparse_baseline: 1;
    uint32_t shared_mixture: 1;
    uint32_t rest: 21;
  } flags;
};

int spqr_matvec(
  // W and meta
  int bits,
  int prob_m,
  int prob_n,
  // Quantization
  int beta1,
  int beta2,
  const void *_raw_data,
  // 32-bit
  void *row_offsets,
  // 16-bit
  void *col_vals,
  int nnz,
  // 16-bit
  // Input
  void *X,
  void *order,
  // Output
  void *y,
  hipStream_t stream,
  void *measurements,
  uint32_t feature_flag) {
  Timer *timer{};
  if (measurements) {
    timer = new Timer(stream);
    timer->start();
  }

  if (prob_m == 0 || prob_n == 0) {
    return 0;
  }

  Features features{._ = feature_flag};

  bool dense_only = (nnz == 0) | features.flags.dense_only;

  const uint64_t *raw_data = (const uint64_t *) _raw_data;
  const half *X_ptr = (const half *) X;
  const int *row_offsets_ptr = (const int *) row_offsets;
  half *y_ptr = (half *) y;
  const auto *col_vals_ptr = (const u32 *) col_vals;
  const short *order_ptr = (const short *) order;

  int ret = 0;


  if (dense_only) {
    if (prob_m % 16 == 0 && prob_n % 256 == 0) {
      CALL_DENSE(spqr_quantized_matvec_dense, 1, 16, 1);
    } else {
      CALL_DENSE(spqr_quantized_matvec_dense, 1, 1, 1);
    }
  } else {
    if (prob_m % 16 == 0 && prob_n % 256 == 0) {
      CALL_FUSED(spqr_quantized_matvec_fused, 1, 16, 2);
    } else if (prob_m % 16 == 0 && prob_n % 128 == 0) {
      CALL_FUSED(spqr_quantized_matvec_fused, 1, 8, 2);
    } else {
      CALL_FUSED(spqr_quantized_matvec_fused, 1, 1, 1);
    }
  }

  if (!features.flags.is_async) {
    CHECK_CUDA(hipDeviceSynchronize());
  }

  if (measurements) {
    static_cast<float *>(measurements)[0] = timer->end();
    delete timer;
  }

  return ret;
}
